#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void
up_sweep(int N, int* device_result, int twod)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int two_dplus1 = 2*twod;

    device_result[(index+1) * two_dplus1 -1] += device_result[index * two_dplus1 + twod -1];
}

__global__ void
down_sweep(int N, int* device_result, int twod)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int two_dplus1 = 2*twod;

    if (((index+1) * two_dplus1 -1) < N) {
        int tmp = device_result[index*two_dplus1+twod-1];
        int tmp1 = device_result[(index+1)*two_dplus1-1];
        device_result[index*two_dplus1+twod-1] = tmp1;
        device_result[(index+1)*two_dplus1-1] = tmp1 + tmp;
    }
    
}

__global__ void
zero(int N, int* device_result)
{
    device_result[N-1] = 0;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    int *tmp = new int[N];
    int d = N/2;
    int twod = 1;
    while (d > THREADS_PER_BLOCK) {
        int block_size = d / THREADS_PER_BLOCK;
        up_sweep<<<block_size, THREADS_PER_BLOCK>>>(N, result, twod);
        d = d / 2;
        twod = twod * 2;
    }

    while (d > 0) {
        up_sweep<<<1, d>>>(N, result, twod);
        d = d / 2;
        twod = twod * 2;
    }

    zero<<<1, 1>>>(N, result);

    // downsweep phase
    d = 1;
    twod = N / 2;
    while (d <= THREADS_PER_BLOCK) {
         down_sweep<<<1, d>>>(N, result, twod);
         twod = twod / 2;
         d = d * 2;
    }

    while (d < N) {
        int block_size = d / THREADS_PER_BLOCK;
        down_sweep<<<block_size, THREADS_PER_BLOCK>>>(N, result, twod);
        twod = twod / 2;
        d = d * 2;
    }

}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    // int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, rounded_length, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    hipFree(device_result);
    hipFree(device_input);
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}
__global__ void
write_flag(int* device_input, int* device_flag, int* device_index, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N-1 && device_input[index] == device_input[index+1]) {
        device_flag[index] = 1;
        device_index[index] = 1;
    }

    if (index < N-1 && device_input[index] != device_input[index+1]) {
        device_flag[index] = 0;
        device_index[index] = 0;
    }

}

__global__ void
print_repeat(int* device_index, int* device_flag, int* device_output, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N && device_flag[index] == 1) {
        device_output[device_index[index]] = index;
    }
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
    int *device_flag;
    int *device_index;
    int *device_count;
    int count;
    int rounded_length = nextPow2(length);

    hipMalloc((void **)&device_flag, rounded_length * sizeof(int));
    hipMalloc((void **)&device_index, rounded_length * sizeof(int));
    hipMalloc((void **)&device_count, 1 * sizeof(int));

    int block_size = length % THREADS_PER_BLOCK !=0 ? length / THREADS_PER_BLOCK+1 : length / THREADS_PER_BLOCK;
    write_flag<<<block_size, THREADS_PER_BLOCK>>>(device_input, device_flag, device_index, length);

    exclusive_scan(device_index, rounded_length, device_index);

    print_repeat<<<block_size, THREADS_PER_BLOCK>>>(device_index, device_flag, device_output, length);

    hipMemcpy(&count, device_index + length -1, 1* sizeof(int), hipMemcpyDeviceToHost);
    hipFree(device_flag);
    hipFree(device_index);
    hipFree(device_count);

    return count; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
